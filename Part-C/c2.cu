#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <stdlib.h>

#define BLOCK_SIZE 32

__constant__ double constFilter[64*3*3*3];

__global__ void Conv2d(const double *I,double* O,int C,int K, int H,int W,int HP,int WP,int FH, int FW)
{
    int k = blockIdx.z;
    int block_row = blockIdx.x;
    int block_col = blockIdx.y;
    int thread_row = threadIdx.x;
    int thread_col = threadIdx.y;

    double OValue = 0;
    __shared__ shared_I[C*(FW + 2*(BLOCK_SIZE - 1)) * (FH + 2*(BLOCK_SIZE - 1))];

    for(int c=0; c<C; ++c) {
        for(int i=0; i<FH + 2*(BLOCK_SIZE - 1); ++i) {
            for(int j=0;j<FW + 2*(BLOCK_SIZE - 1);++j)
                shared_I[c*(FW + 2*(BLOCK_SIZE - 1)) * (FH + 2*(BLOCK_SIZE - 1)) + i*(FW + 2*(BLOCK_SIZE - 1)) + j] = I[c*(HP*WP) + ];
        }
    }

    __syncthreads();
    for(int c=0; c<C; ++c) {
        for(int i = 0; i<FH; ++i) {
            for(int j=0; j<FW; ++j) {
                OValue += I[c*(HP*WP) + block_row*WP*BLOCK_SIZE + block_col*BLOCK_SIZE + (thread_row+i)*WP + thread_col+j] * constFilter[k*(C*FH*FW) + c*(FH*FW) + (FH-i)*FW + (FW-j)];
            }
        }
    }

    O[k*(H*W) + block_row*W*BLOCK_SIZE + block_col*BLOCK_SIZE + thread_row*W + thread_col] = OValue;
}   

double getCheckSum(const double *h_O, int K, int H, int W) {
    double sum = 0.0;
    for (int k = 0; k < K; ++k) {
        for (int i = 0; i < H; ++i) {
            for (int j = 0; j < W; ++j) {
                sum += h_O[k * H * W + i * W + j];
            }
        }
    }
    return sum;
}

int main(int argc,char *argv[]) {
    int H=1024,W=1024,C=3,FW=3,FH=3,K = 64;
    int P = 1;
    int HP = H + 2*P;
    int WP = W + 2*P;

    double *I,*F,*O;
    double *d_I,*d_O;
    int size = C*(H + 2*P)*(W + 2*P);
    int o_size = K*H*W;
    int k_size = K*C*FH*FW;
    I = (double*)malloc(size*sizeof(double));
    O = (double*)malloc(o_size*sizeof(double));
    F = (double*)malloc(k_size*sizeof(double));

    // init matrix
    for(int c=0; c<C; ++c) {
        for(int x=0; x<HP; ++x) {
            for(int y=0; y<WP; ++y) {
                if (x == 0 || x == HP-1 || y==0 || y == WP-1)
                    I[c*(HP*WP) + x*WP + y] = 0;
                else
                    I[c*(HP*WP) + x*WP + y] = c*(x+y);
            }
        }
    }
    printf("INIT MATRIX DONE\n");

    //init kernel
    for(int k=0; k<K; ++k) {
        for(int c=0; c<C; ++c) {
            for(int x=0; x<FH; ++x) {
                for(int y=0; y<FW; ++y) {
                    F[k*(C*FH*FW) + c*(FH*FW) + x*FW + y] = (c+k)*(x+y);
                }
            }
        }
    }
    printf("INIT KERNEL DONE\n");

    if (hipMalloc((void**)&d_I,size*sizeof(double)) != hipSuccess) {
        printf("Failed to allocate GPU memory to I\n");
        exit(0);
    }
    if(hipMalloc((void**)&d_O,o_size*sizeof(double)) != hipSuccess) {
        printf("Failed to allocate GPU memory to O\n");
        exit(0);
    }

    if(hipMemcpy(d_I,I,size*sizeof(double),hipMemcpyHostToDevice) != hipSuccess) {
        printf("Failed to copy I from host to device\n");
        exit(0);
    }
    if (hipMemcpyToSymbol(HIP_SYMBOL(constFilter), F, k_size*sizeof(double)) != hipSuccess) {
        printf("Failed to copy filter to constant memory\n");
        exit(1);
    }
    // kernel code
    dim3 gridDim((H + BLOCK_SIZE -1)/BLOCK_SIZE, (W + BLOCK_SIZE - 1)/BLOCK_SIZE,K);
    dim3 blockDim(BLOCK_SIZE,BLOCK_SIZE);
    Conv2d<<<gridDim,blockDim>>>(d_I,d_O,C,K,H,W,HP,WP,FH,FW);
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if(err != hipSuccess) {
        printf("Error: %s\n",hipGetErrorString(err));
        exit(0);
    }

    if(hipMemcpy(O,d_O,o_size*sizeof(double),hipMemcpyDeviceToHost) != hipSuccess) {
        printf("Failed to copy O from device to host\n");
        exit(0);
    }
    double checksum = getCheckSum(O,K,H,W);
    printf("Checksum: %.3lf\n",checksum);

    hipFree(d_I);
    hipFree(d_O);

    free(I);
    free(O);
    free(F);
    return 0;
}