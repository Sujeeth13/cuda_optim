
#include <hip/hip_runtime.h>
///
/// vecAddKernel00.cu
/// For COMS E6998 Spring 2023
/// Instructor: Parajit Dube and Kaoutar El Maghraoui
/// Based on code from the CUDA Programming Guide
/// By David Newman
/// Created: 2011-02-16
/// Last Modified: 2011-02-16 DVN
///
/// This Kernel adds two Vectors A and B in C on GPU
/// without using coalesced memory access.
/// 

__global__ void AddVectors(const float* A, const float* B, float* C, int N)
{
    int blockStartIndex  = blockIdx.x * blockDim.x * N;
    int threadStartIndex = blockStartIndex + (threadIdx.x * N);
    int threadEndIndex   = threadStartIndex + N;
    int i;

    for( i=threadStartIndex; i<threadEndIndex; ++i ){
        C[i] = A[i] + B[i];
    }
}
